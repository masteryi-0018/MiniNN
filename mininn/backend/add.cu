
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_GPU() {
    printf("hello from GPU \n");
}

int main(void) {
    hello_from_GPU<<<4, 4>>>();
    hipDeviceSynchronize();
    return 0;
}