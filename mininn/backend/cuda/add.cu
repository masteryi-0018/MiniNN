#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

// #include <helper_cuda.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const float* A, const float* B, float* C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i] + 0.0f;
    }
}

void cuda_add_wrapper(const float* h_A, const float* h_B, float* h_C, int numElements) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    size_t size = numElements * sizeof(float);

    // Allocate the device input vector A
    float* d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float* d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float* d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in device memory
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy vector A from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy vector B from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector in host memory.
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy vector C from device to host (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}